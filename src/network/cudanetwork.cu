#include "hip/hip_runtime.h"
#define DEVICE_CODE

#include <iostream>
#include <vector>

#include "network.h"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>

//--------------------------------------------------------------------------------
//---
//--- MACROS
//---
//--------------------------------------------------------------------------------
#define errif( STMT, MSG... ) if( STMT ) { fprintf(stderr, "[%s:%d] '%s' ", __FILE__, __LINE__, #STMT); fprintf(stderr, MSG); fprintf(stderr, "\n"); abort(); }
#define require( STMT ) if( !(STMT) ) { fprintf(stderr, "ASSERTION ERROR! [%s:%d] '%s'\n", __FILE__, __LINE__, #STMT); abort(); }
#define panic() { fprintf(stderr, "PANIC! [%s:%d]\n", __FILE__, __LINE__); abort(); }
#define trap(msg) {std::cerr << __FILE__ << ":" << __LINE__ << ": " << msg << std::endl; abort();}

#define p(msg) std::cout << "[cuda]: " << msg << std::endl

// Number of threads cannot exceed max value of ActivationPartition's offset and
// len fields. If they are of type uchar, then Threads_Per_Block must be < 256
#define Threads_Per_Block 32

// Use no more than 256 bytes of local memory for links
#define Max_Links_Per_Thread (256 / sizeof(CudaLink))
#define Max_Links (Max_Links_Per_Thread * Threads_Per_Block)

#define xcuda(stmt) {                                                   \
        hipError_t err = stmt;                                         \
        if (err != hipSuccess) {                                       \
            std::cerr << __FILE__ << ":" << __LINE__ << ": Failed to run " << #stmt << ". Reason: " << hipGetErrorString(err) << std::endl; \
            abort();                                                    \
        }                                                               \
    }

namespace NEAT {
/*
    __global__ void activate(GpuState *states,
                             RawBuffers bufs,
                             CudaBatchSensorsDims sensor_dims,
                             uint ncycles);
*/

    static uchar *alloc_host(uint size) {
        uchar *result;
        xcuda( hipHostMalloc((void **)&result, size) );
        return result;
    }
    static uchar *alloc_dev(uint size) {
        uchar *result;
        xcuda( hipMalloc((void **)&result, size) );
        return result;
    }
    static void free_host(__inout uchar *&buf, bool tolerate_shutdown = false) {
        if(buf) {
            hipError_t err = hipHostFree(buf);
            if( (err == hipSuccess)
                || (tolerate_shutdown && (err == hipErrorDeinitialized)) ) {
                buf = 0;
            } else {
                std::cerr << "Failed freeing cuda host buffer" << std::endl;
                abort();
            }
        }
    }
    static void free_dev(__inout uchar *&buf) {
        if(buf) {
            xcuda( hipFree(buf) );
            buf = 0;
        }
    }
    static void grow_buffers(__inout uchar *&h_buf, __inout uchar *&d_buf,
                             __inout uint &capacity, __in uint newlen) {
        free_host(h_buf);
        free_dev(d_buf);
        capacity = newlen;
        h_buf = alloc_host(newlen);
        d_buf = alloc_dev(newlen);
    }

    __dh_util CudaLink *links(const RawBuffers &bufs,
                              const Offsets &offs) {
        return (CudaLink *)(bufs.main + offs.main.links);
    }

    __dh_util ActivationPartition *partitions(const RawBuffers &bufs,
                                              const Offsets &offs) {
        return (ActivationPartition *)(bufs.main + offs.main.partitions);
    }

#undef __dh_util

//--------------------------------------------------------------------------------
//---
//--- CLASS CudaNetwork
//---
//--------------------------------------------------------------------------------
    void CudaNetwork::configure_batch(const RawBuffers &bufs_,
                                      const Offsets &offsets_) {
        bufs = bufs_;
        offsets = offsets_;

        memcpy( NEAT::links(bufs, offsets),
                gpu_links.data(),
                sizeof(CudaLink) * gpu_links.size() );
        memcpy( NEAT::partitions(bufs, offsets),
                partitions.data(),
                sizeof(ActivationPartition) * partitions.size() );
    }

    void CudaNetwork::configure(const NetDims &dims_,
                                NetNode *nodes,
                                NetLink *links) {

        static_cast<NetDims &>(dims) = dims_;

        require(dims.nlinks < Max_Links);

        partitions.clear();
        gpu_links.resize(dims.nlinks);

        if(dims.nlinks != 0) {
            ActivationPartition partition;

            for(link_size_t i = 0; i < dims.nlinks; i++) {
                NetLink &link = links[i];
                if( (i % Threads_Per_Block == 0)
                    || (link.out_node_index != partition.out_node_index) ) {

                    if(i != 0) {
                        partitions.push_back(partition);
                    }

                    partition.out_node_index = link.out_node_index;
                    partition.offset = i % Threads_Per_Block;
                    partition.len = 0;
                }
                partition.len++;

                CudaLink &gpu_link = gpu_links[i];
                gpu_link.in_node_index = link.in_node_index;
                gpu_link.partition = partitions.size();
                gpu_link.weight = link.weight;
            }

            partitions.push_back(partition);
        }
        dims.npartitions = partitions.size();
    }

#if false

//--------------------------------------------------------------------------------
//---
//--- CLASS CudaNetworkBatch
//---
//--------------------------------------------------------------------------------
    CudaNetworkBatch::CudaNetworkBatch(int device_, uint nnets_)
        : device(device_), nnets(nnets_) {

        hipSetDevice(device);

        memset(&h_bufs, 0, sizeof(h_bufs));
        memset(&d_bufs, 0, sizeof(d_bufs));
        memset(&offsets, 0, sizeof(offsets));
        memset(&capacity, 0, sizeof(capacity));
        memset(&lens, 0, sizeof(lens));

        h_gpu_states = (GpuState *)alloc_host(sizeof(GpuState) * nnets);
        d_gpu_states = (GpuState *)alloc_dev(sizeof(GpuState) * nnets);
    }

    CudaNetworkBatch::~CudaNetworkBatch() {
        hipSetDevice(device);

        free_host((uchar *&)h_gpu_states);
        free_dev((uchar *&)d_gpu_states);

        free_host(h_bufs.main);
        // the host input buffer is in the sensors object
        free_host(h_bufs.output);

        free_dev(d_bufs.main);
        free_dev(d_bufs.input);
        free_dev(d_bufs.output);
    }

    void CudaNetworkBatch::configure(CudaBatchSensors *batch_sensors,
                                     CudaNetwork **nets,
                                     uint nnets) {
        assert(nnets = this->nnets);

        hipSetDevice(device);

        sensor_dims = batch_sensors->get_dims();
        memset(&lens, 0, sizeof(lens));
        sizeof_shared = 0;
        istep_output = 0;

        Offsets nets_offs[nnets];

        for(uint i = 0; i < nnets; i++) {
            CudaNetwork &net = *nets[i];
            CudaNetDims &dims = net.dims;

            Lens net_lens;
            Offsets &net_offs = nets_offs[i];
            uint net_sizeof_shared =
                (2 * sizeof(real_t) * dims.nnodes.all)
                + (sizeof(real_t) * Threads_Per_Block);

            //main buffer
            {
                uint sizeof_links = sizeof(CudaLink) * dims.nlinks;
                uint sizeof_partitions = sizeof(ActivationPartition) * dims.npartitions;

                net_lens.main = sizeof_links + sizeof_partitions;

                net_offs.main.links = this->lens.main;
                net_offs.main.partitions = net_offs.main.links + sizeof_links;

                lens.main += net_lens.main;
            }

            //input buffer
            {
                uint sizeof_step_parms = StepParms::sizeof_buf(sensor_dims);

                net_lens.input = sizeof_step_parms;

                //If all nets share the same input.
                {
                    assert(sizeof_step_parms == batch_sensors->sizeof_buf());
                    net_offs.input.step_parms = 0;
                    lens.input = max(lens.input, net_lens.input);
                }
            }

            //output buffer
            {
                uint sizeof_activation = 
                    sizeof(real_t) * dims.nnodes.output * sensor_dims.nsteps;

                net_lens.output = sizeof_activation;

                net_offs.output.activation = this->lens.output;
                // gpu requires proper alignment
                assert(net_offs.output.activation % sizeof(real_t) == 0);

                lens.output += net_lens.output;
            }

            sizeof_shared = max(sizeof_shared, net_sizeof_shared);
            
            net.offsets = net_offs;
        }

        if(lens.main > capacity.main) {
            uint newlen = uint(lens.main * 1.4);
            p("alloc main: " << newlen);
            grow_buffers(h_bufs.main, d_bufs.main, capacity.main, newlen);
        }
        if(lens.input > capacity.input) {
            uint newlen = uint(lens.input);
            p("alloc input: " << newlen);
            assert(capacity.input == 0); // should only alloc once
            d_bufs.input = alloc_dev(newlen);
            capacity.input = newlen;
        }
        if(lens.output > capacity.output) {
            uint newlen = uint(lens.output);
            p("alloc output: " << newlen);
            assert(capacity.output == 0); // should only alloc once
            grow_buffers(h_bufs.output, d_bufs.output, capacity.output, newlen);
        }

        for(uint i = 0; i < nnets; i++) {
            CudaNetwork *net = nets[i];
            net->configure_batch(h_bufs, nets_offs[i], &istep_output);

            GpuState &gpu = h_gpu_states[i];
            gpu.dims = net->dims;
            gpu.offsets = net->offsets;
        }

        xcuda( hipMemcpy(d_bufs.input, batch_sensors->get_h_buf(), lens.input, hipMemcpyHostToDevice) );
        xcuda( hipMemcpy(d_gpu_states, h_gpu_states, sizeof(GpuState) * nnets, hipMemcpyHostToDevice) );
        xcuda( hipMemcpy(d_bufs.main, h_bufs.main, lens.main, hipMemcpyHostToDevice) );
    }

    void CudaNetworkBatch::activate(uint ncycles) {
        NEAT::activate<<<nnets, Threads_Per_Block, sizeof_shared>>>(d_gpu_states,
                                                                    d_bufs,
                                                                    sensor_dims,
                                                                    ncycles);
        
        xcuda( hipMemcpy(h_bufs.output,
                          d_bufs.output,
                          lens.output,
                          hipMemcpyDeviceToHost) );
    }

    void CudaNetworkBatch::set_output_step(size_t istep) {
        istep_output = istep;
    }

//--------------------------------------------------------------------------------
//---
//--- CLASS CudaBatchSensors
//---
//--------------------------------------------------------------------------------
    CudaBatchSensors::CudaBatchSensors(const CudaBatchSensorsDims &dims_)
        : dims(dims_) {

        h_buf = alloc_host(sizeof_buf());
    }

    CudaBatchSensors::~CudaBatchSensors() {
        free_host(h_buf, true);
    }

    void CudaBatchSensors::configure_step(size_t istep,
                                          const std::vector<real_t> &values,
                                          bool clear_noninput) {
        assert(values.size() == dims.nsensors);
        if(istep == 0) {
            require(clear_noninput);
        }

        StepParms *parms = StepParms::get(h_buf, dims, istep);

        parms->clear_noninput = clear_noninput;
        for(size_t i = 0; i < dims.nsensors; i++)
            parms->activations[i] = values[i];
    }

//--------------------------------------------------------------------------------
//---
//--- GPU KERNEL CODE
//---
//--------------------------------------------------------------------------------
    __device__ void sum_partition(float *x, int i, int n, float *result) {
        int stride = __popc(n) == 1 ? n >> 1 : 1 << 31 - __clz(n);

        if( (stride > 0) && (i + stride < n) ) {
            x[i] += x[i + stride];
        }
      
        __syncthreads();

        stride >>= 1;
        // max_stride necessary to keep all threads from all partitions in sync.
        for(int max_stride = Threads_Per_Block >> 1; max_stride > 0; stride >>= 1, max_stride >>= 1) {
            if(i < stride) {
                x[i] += x[i + stride];
            }
            __syncthreads();
        }

        if(i == 0) {
            *result += x[0];
        }

        __syncthreads();
    }

    __global__ void test_sum_partition_kernel(float *x, int n, float *result) {
        uint tid = threadIdx.x;
        __shared__ float shx[Threads_Per_Block];
        *result = 0;
        int i;
        if(tid < n) {
            shx[tid] = x[tid];
            i = tid;
        } else {
            i = 1; n = 0;
        }
        sum_partition(shx, i, n, result);
    }

    inline __device__ real_t fsigmoid(real_t activesum,real_t slope,real_t constant) {
        //NON-SHIFTED STEEPENED
        return (1/(1+(exp(-(slope*activesum))))); //Compressed
    }

    __global__ void activate(GpuState *states,
                             RawBuffers bufs,
                             CudaBatchSensorsDims sensor_dims,
                             uint ncycles) {
        // to print sensors:
        // p *(@global float * @local)(real_t *foo)@sensor_dims.nsensors

        GpuState state = states[blockIdx.x];
        uint tid = threadIdx.x;

        //---
        //--- Config shared memory
        //---
        extern __shared__ char __shared_buf[];

        // in cuda-gdb: print *((@shared float*)activation + i)
        //              print *((@shared float*)newactivation)@6
        real_t *activation = (real_t *)__shared_buf;
        real_t *newactivation = activation + state.dims.nnodes.all;
        real_t *partial_activation = newactivation + state.dims.nnodes.all;


        //---
        //--- Cache link/partitions in local memory.
        //---
        const int ncycle_its = 1 + (state.dims.nlinks - 1) / Threads_Per_Block;

        CudaLink local_links[Max_Links_Per_Thread];
        ActivationPartition local_partitions[Max_Links_Per_Thread];
        for(uint ilink = tid, it = 0; it < ncycle_its; ilink += Threads_Per_Block, it++) {
            CudaLink &link = local_links[it];
            ActivationPartition &p = local_partitions[it];
            if(ilink < state.dims.nlinks) {
                link = links(bufs, state.offsets)[ilink];
                p = partitions(bufs, state.offsets)[local_links[it].partition];
            }
        }


        //---
        //--- Process all batch steps
        //---
        for(size_t istep = 0; istep < sensor_dims.nsteps; istep++) {
            StepParms *parms = step_parms(bufs, state.offsets, sensor_dims, istep);

            //---
            //--- Load step activations
            //---
            for(uint inode = tid; inode < state.dims.nnodes.all; inode += Threads_Per_Block) {
                if(inode < state.dims.nnodes.input) {
                    //---
                    //--- Bias/Sensor node
                    //---
                    const uint nbias = state.dims.nnodes.bias;
                    if(inode < nbias) {
                        activation[inode] = 1.0;
                    } else {
                        activation[inode] =
                            parms->activations[inode - nbias];
                    }
                    newactivation[inode] = activation[inode];
                } else {
                    //---
                    //--- Output/Hidden node
                    //---
                    if( parms->clear_noninput ) {
                        activation[inode] = 0.0;
                    }
                }
            }
            __syncthreads();

            //---
            //--- For each cycle of this step.
            //---
            for(uint icycle = 0; icycle < ncycles; icycle++) {

                //---
                //--- Reset new activation noninput
                //---
                for(uint inode = tid + state.dims.nnodes.input;
                    inode < state.dims.nnodes.all;
                    inode += Threads_Per_Block) {

                    newactivation[inode] = 0.0;
                }

                //---
                //--- Compute new activation sums
                //---
                for(uint ilink = tid, it = 0; it < ncycle_its; ilink += Threads_Per_Block, it++) {
                    float *partition_x;
                    int partition_i;
                    int partition_n;
                    float *result;

                    if(ilink < state.dims.nlinks) {
                        CudaLink &link = local_links[it];
                        partial_activation[tid] = link.weight * activation[link.in_node_index];

                        ActivationPartition &p = local_partitions[it];
                        partition_x = partial_activation + p.offset;
                        partition_i = tid - p.offset;
                        partition_n = p.len;
                        result = newactivation + p.out_node_index;
                    } else {
                        partition_x = NULL;
                        partition_i = 1;
                        partition_n = 0;
                        result = NULL;
                    }

                    __syncthreads();

                    sum_partition(partition_x,
                                  partition_i,
                                  partition_n,
                                  result);
                }

                //---
                //--- Compute new activations from sums
                //---
                for(uint inode = tid + state.dims.nnodes.input;
                    inode < state.dims.nnodes.all;
                    inode += Threads_Per_Block) {

                    newactivation[inode] = fsigmoid(newactivation[inode],
                                                    4.924273,
                                                    2.4621365);
                }
                __syncthreads();

                //---
                //--- "activation" now the current state.
                //---
                {
                    float *swap = newactivation;
                    newactivation = activation;
                    activation = swap;
                }
            } // end of cycle

            //---
            //--- Save step output to global memory
            //---
            for(uint i = tid; i < state.dims.nnodes.output; i += Threads_Per_Block) {
                output_activations(bufs, state.offsets, state.dims, istep)[i] = 
                    activation[state.dims.nnodes.input + i];
            }
        } // end of step
    }

    void test_sum_partition() {
        for(size_t n = 1; n <= Threads_Per_Block; n++) {
            real_t x[n];
            size_t sizeof_x = sizeof(real_t) * n;

            real_t expected = 0.0;
            for(size_t i = 0; i < n; i++) {
                //x[i] = real_t(i) + 1;
                x[i] = drand48();
                expected += x[i];
            }

            real_t actual = -100;

            real_t *d_x = (real_t *)alloc_dev(sizeof_x);
            xcuda( hipMemcpy(d_x,
                              x,
                              sizeof_x,
                              hipMemcpyHostToDevice) );

            real_t *d_actual = (real_t *)alloc_dev(sizeof(real_t));

            NEAT::test_sum_partition_kernel<<<1, Threads_Per_Block>>>(d_x, n, d_actual);
            
            xcuda( hipMemcpy(&actual,
                              d_actual,
                              sizeof(real_t),
                              hipMemcpyDeviceToHost) );

            if( fabs(expected - actual) / expected >= 0.05 ) {
                std::cout << "n=" << n << ", Expected=" << expected << ", Actual=" << actual << std::endl;
            }
        }
            
        exit(0);
    }
#endif // false

} // namespace NEAT

