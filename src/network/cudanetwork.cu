#include "hip/hip_runtime.h"
#define DEVICE_CODE

#include <iostream>
#include <vector>

#include "cudanetwork.h"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>

//--------------------------------------------------------------------------------
//---
//--- MACROS
//---
//--------------------------------------------------------------------------------
#define errif( STMT, MSG... ) if( STMT ) { fprintf(stderr, "[%s:%d] '%s' ", __FILE__, __LINE__, #STMT); fprintf(stderr, MSG); fprintf(stderr, "\n"); abort(); }
#define require( STMT ) if( !(STMT) ) { fprintf(stderr, "ASSERTION ERROR! [%s:%d] '%s'\n", __FILE__, __LINE__, #STMT); abort(); }
#define panic() { fprintf(stderr, "PANIC! [%s:%d]\n", __FILE__, __LINE__); abort(); }
#define trap(msg) {std::cerr << __FILE__ << ":" << __LINE__ << ": " << msg << std::endl; abort();}

#define p(msg) std::cout << "[cuda]: " << msg << std::endl

// Number of threads cannot exceed max value of ActivationPartition's offset and
// len fields. If they are of type uchar, then Threads_Per_Block must be < 256
#define Threads_Per_Block 32

// Use no more than 256 bytes of local memory for links
#define Max_Links_Per_Thread (256 / sizeof(CudaLink))
#define Max_Links (Max_Links_Per_Thread * Threads_Per_Block)

#define xcuda(stmt) {                                                   \
        hipError_t err = stmt;                                         \
        if (err != hipSuccess) {                                       \
            std::cerr << __FILE__ << ":" << __LINE__ << ": Failed to run " << #stmt << ". Reason: " << hipGetErrorString(err) << std::endl; \
            exit(1);                                                    \
        }                                                               \
    }

namespace NEAT {
    __global__ void activate(GpuState *states,
                             RawBuffers bufs,
                             uint ncycles);

    static uchar *alloc_host(uint size) {
        uchar *result;
        xcuda( hipHostMalloc((void **)&result, size) );
        return result;
    }
    static uchar *alloc_dev(uint size) {
        uchar *result;
        xcuda( hipMalloc((void **)&result, size) );
        return result;
    }
    static void free_host(__inout uchar *&buf) {
        if(buf) {
            xcuda( hipHostFree(buf) );
            buf = 0;
        }
    }
    static void free_dev(__inout uchar *&buf) {
        if(buf) {
            xcuda( hipFree(buf) );
            buf = 0;
        }
    }
    static void grow_buffers(__inout uchar *&h_buf, __inout uchar *&d_buf,
                             __inout uint &capacity, __in uint newlen) {
        free_host(h_buf);
        free_dev(d_buf);
        capacity = newlen;
        h_buf = alloc_host(newlen);
        d_buf = alloc_dev(newlen);
    }

#define __dh_util static inline __device__ __host__

    __dh_util CudaLink *links(const RawBuffers &bufs,
                              const Offsets &offs) {
        return (CudaLink *)(bufs.main + offs.main.links);
    }

    __dh_util ActivationPartition *partitions(const RawBuffers &bufs,
                                              const Offsets &offs) {
        return (ActivationPartition *)(bufs.main + offs.main.partitions);
    }

    __dh_util real_t *sensor_activations(const RawBuffers &bufs,
                                        const Offsets &offs) {
        return (real_t *)(bufs.input + offs.input.activation);
    }

    __dh_util real_t *noninput_activations(const RawBuffers &bufs,
                                           const Offsets &offs) {
        return (real_t *)(bufs.main + offs.main.activation);
    }

    __dh_util real_t *output_activations(const RawBuffers &bufs,
                                         const Offsets &offs) {
        return (real_t *)(bufs.output + offs.output.activation);
    }

    __dh_util ActivateParms &activate_parms(const RawBuffers &bufs,
                                            const Offsets &offs) {
        return *(ActivateParms *)(bufs.input + offs.input.activate_parms);
    }

#undef __dh_util

//--------------------------------------------------------------------------------
//---
//--- CLASS CudaNetworkBatch
//---
//--------------------------------------------------------------------------------
    CudaNetworkBatch::CudaNetworkBatch(uint nnets_)
        : nnets(nnets_) {
        memset(&h_bufs, 0, sizeof(h_bufs));
        memset(&d_bufs, 0, sizeof(d_bufs));
        memset(&offsets, 0, sizeof(offsets));
        memset(&capacity, 0, sizeof(capacity));
        memset(&lens, 0, sizeof(lens));

        h_gpu_states = (GpuState *)alloc_host(sizeof(GpuState) * nnets);
        d_gpu_states = (GpuState *)alloc_dev(sizeof(GpuState) * nnets);
    }

    CudaNetworkBatch::~CudaNetworkBatch() {
        free_host((uchar *&)h_gpu_states);
        free_dev((uchar *&)d_gpu_states);

        free_host(h_bufs.main);
        free_host(h_bufs.input);
        free_host(h_bufs.output);

        free_dev(d_bufs.main);
        free_dev(d_bufs.input);
        free_dev(d_bufs.output);
    }

    void CudaNetworkBatch::configure(CudaNetwork **nets,
                                     uint nnets) {
        assert(nnets = this->nnets);

        memset(&lens, 0, sizeof(lens));
        sizeof_shared = 0;

        Offsets nets_offs[nnets];

        for(uint i = 0; i < nnets; i++) {
            CudaNetwork &net = *nets[i];
            CudaNetDims &dims = net.dims;

            Lens net_lens;
            Offsets &net_offs = nets_offs[i];
            uint net_sizeof_shared =
                (2 * sizeof(real_t) * dims.nnodes.all)
                + (sizeof(real_t) * Threads_Per_Block);

            //main buffer
            {
                uint sizeof_activation = sizeof(real_t) * (dims.nnodes.noninput);
                uint sizeof_links = sizeof(CudaLink) * dims.nlinks;
                uint sizeof_partitions = sizeof(ActivationPartition) * dims.npartitions;

                net_lens.main = sizeof_activation + sizeof_links + sizeof_partitions;

                net_offs.main.activation = this->lens.main;
                // gpu requires proper alignment
                assert(net_offs.main.activation % sizeof(real_t) == 0);
                net_offs.main.links = net_offs.main.activation + sizeof_activation;
                net_offs.main.partitions = net_offs.main.links + sizeof_links;
            }

            //input buffer
            {
                uint sizeof_parms = sizeof(ActivateParms);
                uint sizeof_activation = sizeof(real_t) * dims.nnodes.sensor;

                net_lens.input = sizeof_parms + sizeof_activation;

                net_offs.input.activate_parms = this->lens.input;
                net_offs.input.activation = net_offs.input.activate_parms + sizeof_parms;
                // gpu requires proper alignment
                assert(net_offs.input.activation % sizeof(real_t) == 0);
            }

            //output buffer
            {
                uint sizeof_activation = sizeof(real_t) * dims.nnodes.output;

                net_lens.output = sizeof_activation;

                net_offs.output.activation = this->lens.output;
                // gpu requires proper alignment
                assert(net_offs.output.activation % sizeof(real_t) == 0);
            }

            sizeof_shared = max(sizeof_shared, net_sizeof_shared);
            
            net.offsets = net_offs;
            lens += net_lens;
        }

        if(lens.main > capacity.main) {
            uint newlen = uint(lens.main * 1.4);
            p("alloc main: " << newlen);
            grow_buffers(h_bufs.main, d_bufs.main, capacity.main, newlen);
        }
        if(lens.input > capacity.input) {
            uint newlen = uint(lens.input);
            p("alloc input: " << newlen);
            assert(capacity.input == 0); // should only alloc once
            grow_buffers(h_bufs.input, d_bufs.input, capacity.input, newlen);
        }
        if(lens.output > capacity.output) {
            uint newlen = uint(lens.output);
            p("alloc output: " << newlen);
            assert(capacity.output == 0); // should only alloc once
            grow_buffers(h_bufs.output, d_bufs.output, capacity.output, newlen);
        }

        for(uint i = 0; i < nnets; i++) {
            CudaNetwork *net = nets[i];
            net->set_bufs(h_bufs, nets_offs[i]);

            GpuState &gpu = h_gpu_states[i];
            gpu.dims = net->dims;
            gpu.offsets = net->offsets;
        }

        xcuda( hipMemcpy(d_gpu_states, h_gpu_states, sizeof(GpuState) * nnets, hipMemcpyHostToDevice) );
        xcuda( hipMemcpy(d_bufs.main, h_bufs.main, lens.main, hipMemcpyHostToDevice) );
    }

    void CudaNetworkBatch::activate(uint ncycles) {
        xcuda( hipMemcpy(d_bufs.input,
                          h_bufs.input,
                          lens.input,
                          hipMemcpyHostToDevice) );

        NEAT::activate<<<nnets, Threads_Per_Block, sizeof_shared>>>(d_gpu_states,
                                                                    d_bufs,
                                                                    ncycles);
        
        xcuda( hipMemcpy(h_bufs.output,
                          d_bufs.output,
                          lens.output,
                          hipMemcpyDeviceToHost) );
    }

    std::vector<real_t> &CudaNetworkBatch::get_activations(CudaNetwork *net,
                                                           __out std::vector<real_t> &result) {
        result.clear();
        for(size_t i = 0; i < net->dims.nnodes.bias; i++) {
            result.push_back(1.0);
        }
        for(size_t i = 0; i < net->dims.nnodes.sensor; i++) {
            result.push_back(sensor_activations(h_bufs, net->offsets)[i]);
        }

        real_t noninput[net->dims.nnodes.noninput];
        xcuda( hipMemcpy(noninput, noninput_activations(d_bufs, net->offsets), sizeof(noninput), hipMemcpyDeviceToHost) );

        for(size_t i = 0; i < net->dims.nnodes.noninput; i++) {
            result.push_back(noninput[i]);
        }

        return result;
    }

//--------------------------------------------------------------------------------
//---
//--- CLASS CudaNetwork
//---
//--------------------------------------------------------------------------------
    void CudaNetwork::set_bufs(const RawBuffers &bufs_,
                               const Offsets &offsets_) {
        bufs = bufs_;
        offsets = offsets_;

        memcpy( NEAT::links(bufs, offsets),
                gpu_links.data(),
                sizeof(CudaLink) * gpu_links.size() );
        memcpy( NEAT::partitions(bufs, offsets),
                partitions.data(),
                sizeof(ActivationPartition) * partitions.size() );

        activate_parms(bufs, offsets).clear_noninput = true;
        activate_parms(bufs, offsets).enabled = true;
    }

    void CudaNetwork::set_clear_noninput(bool val) {
        activate_parms(bufs, offsets).clear_noninput = val;
    }

    void CudaNetwork::disable() {
        activate_parms(bufs, offsets).enabled = false;
    }

    bool CudaNetwork::is_enabled() {
        return activate_parms(bufs, offsets).enabled;
    }

    void CudaNetwork::configure(const NetDims &dims_,
                                NetNode *nodes,
                                NetLink *links) {

        static_cast<NetDims &>(dims) = dims_;

        require(dims.nlinks < Max_Links);

        partitions.clear();
        gpu_links.resize(dims.nlinks);

        if(dims.nlinks != 0) {
            ActivationPartition partition;

            for(link_size_t i = 0; i < dims.nlinks; i++) {
                NetLink &link = links[i];
                if( (i % Threads_Per_Block == 0)
                    || (link.out_node_index != partition.out_node_index) ) {

                    if(i != 0) {
                        partitions.push_back(partition);
                    }

                    partition.out_node_index = link.out_node_index;
                    partition.offset = i % Threads_Per_Block;
                    partition.len = 0;
                }
                partition.len++;

                CudaLink &gpu_link = gpu_links[i];
                gpu_link.in_node_index = link.in_node_index;
                gpu_link.partition = partitions.size();
                gpu_link.weight = link.weight;
            }

            partitions.push_back(partition);
        }
        dims.npartitions = partitions.size();
    }

    void CudaNetwork::load_sensors(const std::vector<real_t> &sensvals,
                                   bool clear_noninput) {
        memcpy( sensor_activations(bufs, offsets),
                sensvals.data(),
                sizeof(real_t) * dims.nnodes.sensor );

        activate_parms(bufs, offsets).clear_noninput = clear_noninput;
    }

    real_t CudaNetwork::get_output(size_t index) {
        return output_activations(bufs, offsets)[index];
    }

//--------------------------------------------------------------------------------
//---
//--- GPU KERNEL CODE
//---
//--------------------------------------------------------------------------------
    __device__ void sum_partition(float *x, int i, int n, float *result) {
        int stride = __popc(n) == 1 ? n >> 1 : 1 << 31 - __clz(n);

        if( (stride > 0) && (i + stride < n) ) {
            x[i] += x[i + stride];
        }
      
        __syncthreads();

        stride >>= 1;
        // max_stride necessary to keep all threads from all partitions in sync.
        for(int max_stride = Threads_Per_Block >> 1; max_stride > 0; stride >>= 1, max_stride >>= 1) {
            if(i < stride) {
                x[i] += x[i + stride];
            }
            __syncthreads();
        }

        if(i == 0) {
            *result += x[0];
        }

        __syncthreads();
    }

    __global__ void test_sum_partition_kernel(float *x, int n, float *result) {
        uint tid = threadIdx.x;
        __shared__ float shx[Threads_Per_Block];
        *result = 0;
        int i;
        if(tid < n) {
            shx[tid] = x[tid];
            i = tid;
        } else {
            i = 1; n = 0;
        }
        sum_partition(shx, i, n, result);
    }

    inline __device__ real_t fsigmoid(real_t activesum,real_t slope,real_t constant) {
        //NON-SHIFTED STEEPENED
        return (1/(1+(exp(-(slope*activesum))))); //Compressed
    }

    __global__ void activate(GpuState *states,
                             RawBuffers bufs,
                             uint ncycles) {
        GpuState state = states[blockIdx.x];
        if(!activate_parms(bufs, state.offsets).enabled) {
            return;
        }
        // to print sensors:
        // p *(@global float * @local)(bufs.input + state.offsets.input.activation)@N

        extern __shared__ char __shared_buf[];

        // in cuda-gdb: print *((@shared float*)activation + i)
        //              print *((@shared float*)newactivation)@6
        real_t *activation = (real_t *)__shared_buf;
        real_t *newactivation = activation + state.dims.nnodes.all;
        real_t *partial_activation = newactivation + state.dims.nnodes.all;

        uint tid = threadIdx.x;

        for(uint inode = tid; inode < state.dims.nnodes.all; inode += Threads_Per_Block) {
            if(inode < state.dims.nnodes.input) {
                const uint nbias = state.dims.nnodes.bias;
                if(inode < nbias) {
                    activation[inode] = 1.0;
                } else {
                    activation[inode] =
                        sensor_activations(bufs, state.offsets)[inode - nbias];
                }
                newactivation[inode] = activation[inode];
            } else {
                if( activate_parms(bufs, state.offsets).clear_noninput ) {
                    activation[inode] = 0.0;
                } else {
                    const uint ninput = state.dims.nnodes.input;
                    activation[inode] =
                        noninput_activations(bufs, state.offsets)[inode - ninput];
                }
            }
        }
        __syncthreads();

        const int nits = 1 + (state.dims.nlinks - 1) / Threads_Per_Block;

        CudaLink local_links[Max_Links_Per_Thread];
        ActivationPartition local_partitions[Max_Links_Per_Thread];
        for(uint ilink = tid, it = 0; it < nits; ilink += Threads_Per_Block, it++) {
            CudaLink &link = local_links[it];
            ActivationPartition &p = local_partitions[it];
            if(ilink < state.dims.nlinks) {
                link = links(bufs, state.offsets)[ilink];
                p = partitions(bufs, state.offsets)[local_links[it].partition];
            }
        }

        for(uint icycle = 0; icycle < ncycles; icycle++) {
            for(uint inode = tid + state.dims.nnodes.input;
                inode < state.dims.nnodes.all;
                inode += Threads_Per_Block) {
                newactivation[inode] = 0.0;
            }

            for(uint ilink = tid, it = 0; it < nits; ilink += Threads_Per_Block, it++) {
                float *partition_x;
                int partition_i;
                int partition_n;
                float *result;

                if(ilink < state.dims.nlinks) {
                    CudaLink &link = local_links[it];
                    partial_activation[tid] = link.weight * activation[link.in_node_index];

                    ActivationPartition &p = local_partitions[it];
                    partition_x = partial_activation + p.offset;
                    partition_i = tid - p.offset;
                    partition_n = p.len;
                    result = newactivation + p.out_node_index;
                } else {
                    partition_x = NULL;
                    partition_i = 1;
                    partition_n = 0;
                    result = NULL;
                }

                __syncthreads();

                sum_partition(partition_x,
                              partition_i,
                              partition_n,
                              result);
            }

            for(uint inode = tid + state.dims.nnodes.input;
                inode < state.dims.nnodes.all;
                inode += Threads_Per_Block) {

                newactivation[inode] = fsigmoid(newactivation[inode],
                                                4.924273,
                                                2.4621365);
            }
            __syncthreads();

            {
                float *swap = newactivation;
                newactivation = activation;
                activation = swap;
            }
        }

        for(uint inode = tid + state.dims.nnodes.input;
            inode < state.dims.nnodes.all;
            inode += Threads_Per_Block) {

            uint offset = inode - state.dims.nnodes.input;
            real_t act = activation[inode];

            noninput_activations(bufs, state.offsets)[offset] = act;

            if(inode < (state.dims.nnodes.input + state.dims.nnodes.output)) {
                output_activations(bufs, state.offsets)[offset] = act;
            }
        }
    }

    void test_sum_partition() {
        for(size_t n = 1; n <= Threads_Per_Block; n++) {
            real_t x[n];
            size_t sizeof_x = sizeof(real_t) * n;

            real_t expected = 0.0;
            for(size_t i = 0; i < n; i++) {
                //x[i] = real_t(i) + 1;
                x[i] = drand48();
                expected += x[i];
            }

            real_t actual = -100;

            real_t *d_x = (real_t *)alloc_dev(sizeof_x);
            xcuda( hipMemcpy(d_x,
                              x,
                              sizeof_x,
                              hipMemcpyHostToDevice) );

            real_t *d_actual = (real_t *)alloc_dev(sizeof(real_t));

            NEAT::test_sum_partition_kernel<<<1, Threads_Per_Block>>>(d_x, n, d_actual);
            
            xcuda( hipMemcpy(&actual,
                              d_actual,
                              sizeof(real_t),
                              hipMemcpyDeviceToHost) );

            if( fabs(expected - actual) / expected >= 0.05 ) {
                std::cout << "n=" << n << ", Expected=" << expected << ", Actual=" << actual << std::endl;
            }
        }
            
        exit(0);
    }

} // namespace NEAT
